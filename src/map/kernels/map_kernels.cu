#include "hip/hip_runtime.h"
/*Copyright(c) 2020, The Regents of the University of California, Davis.            */
/*                                                                                  */
/*                                                                                  */
/*Redistribution and use in source and binary forms, with or without modification,  */
/*are permitted provided that the following conditions are met :                    */
/*                                                                                  */
/*1. Redistributions of source code must retain the above copyright notice, this    */
/*list of conditions and the following disclaimer.                                  */
/*2. Redistributions in binary form must reproduce the above copyright notice,      */
/*this list of conditions and the following disclaimer in the documentation         */
/*and / or other materials provided with the distribution.                          */
/*                                                                                  */
/*THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND   */
/*ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED     */
/*WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.*/
/*IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,  */
/*INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES(INCLUDING, BUT */
/*NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR*/
/*PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, */
/*WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT(INCLUDING NEGLIGENCE OR OTHERWISE) */
/*ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE        */
/*POSSIBILITY OF SUCH DAMAGE.                                                       */
/************************************************************************************/
/************************************************************************************/

#pragma once

#include <cstdint>

namespace GpuBTree {
namespace kernels {
template<typename KeyT, typename ValueT, typename SizeT, typename AllocatorT>
__global__ void insert_keys(uint32_t* d_root,
                            KeyT* d_keys,
                            ValueT* d_values,
                            SizeT num_keys,
                            AllocatorT allocator) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t laneId = threadIdx.x & 0x1F;

  KeyT myKey;
  ValueT myValue;
  bool to_insert = false;

  if ((tid - laneId) >= num_keys)
    return;

  if (tid < num_keys) {
    myKey = d_keys[tid] + 2;
    myValue = d_values[tid] + 2;
    to_insert = true;
  }

  warps::insertion_unit(to_insert, myKey, myValue, d_root, &allocator);
}

template<typename AllocatorT>
__global__ void init_btree(uint32_t* d_root, AllocatorT allocator) {
  uint32_t laneId = threadIdx.x & 0x1F;

  uint32_t root_id;
  if (laneId == 0)
    root_id = allocator.allocate();

  root_id = __shfl_sync(WARP_MASK, root_id, 0);

  *d_root = root_id;
  uint32_t* tree_root = allocator.getAddressPtr(root_id);

  if (laneId < 2)
    tree_root[laneId] = 1 - laneId;
}

template<typename KeyT, typename ValueT, typename SizeT, typename AllocatorT>
__global__ void search_b_tree(uint32_t* d_root,
                              KeyT* d_queries,
                              ValueT* d_results,
                              SizeT num_queries,
                              AllocatorT allocator) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t laneId = threadIdx.x & 0x1F;
  if ((tid - laneId) >= num_queries)
    return;

  uint32_t myQuery = 0;
  uint32_t myResult = SEARCH_NOT_FOUND;
  bool to_search = false;

  if (tid < num_queries) {
    myQuery = d_queries[tid] + 2;
    to_search = true;
  }

  warps::search_unit(to_search, laneId, myQuery, myResult, d_root, &allocator);

  if (tid < num_queries)
    myResult = myResult ? myResult - 2 : myResult;
  d_results[tid] = myResult;
}

};  // namespace kernels
};  // namespace GpuBTree
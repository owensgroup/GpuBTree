/*Copyright(c) 2020, The Regents of the University of California, Davis.            */
/*                                                                                  */
/*                                                                                  */
/*Redistribution and use in source and binary forms, with or without modification,  */
/*are permitted provided that the following conditions are met :                    */
/*                                                                                  */
/*1. Redistributions of source code must retain the above copyright notice, this    */
/*list of conditions and the following disclaimer.                                  */
/*2. Redistributions in binary form must reproduce the above copyright notice,      */
/*this list of conditions and the following disclaimer in the documentation         */
/*and / or other materials provided with the distribution.                          */
/*                                                                                  */
/*THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND   */
/*ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED     */
/*WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.*/
/*IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,  */
/*INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES(INCLUDING, BUT */
/*NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR*/
/*PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, */
/*WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT(INCLUDING NEGLIGENCE OR OTHERWISE) */
/*ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE        */
/*POSSIBILITY OF SUCH DAMAGE.                                                       */
/************************************************************************************/
/************************************************************************************/

#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <random>
#include <vector>

#include "GpuBTree.h"

int main(int argc, char* argv[]) {
  GpuBTree::GpuBTreeMap<uint32_t, uint32_t, uint32_t> btree;

  // Input number of keys
  uint32_t numKeys = 1 << 20;
  if (argc > 1)
    numKeys = std::atoi(argv[1]);

  // Prepare the keys
  std::vector<uint32_t> keys;
  std::vector<uint32_t> values;
  keys.reserve(numKeys);
  values.reserve(numKeys);
  for (int iKey = 0; iKey < numKeys; iKey++) {
    keys.push_back(iKey);
  }

  // shuffle the keys
  std::random_device rd;
  std::mt19937 g(rd());
  std::shuffle(keys.begin(), keys.end(), g);

  // assign the values
  for (int iKey = 0; iKey < numKeys; iKey++) {
    values.push_back(keys[iKey]);
  }

  // Move data to GPU
  uint32_t *d_keys, *d_values;
  CHECK_ERROR(memoryUtil::deviceAlloc(d_keys, numKeys));
  CHECK_ERROR(memoryUtil::deviceAlloc(d_values, numKeys));
  CHECK_ERROR(memoryUtil::cpyToDevice(keys.data(), d_keys, numKeys));
  CHECK_ERROR(memoryUtil::cpyToDevice(values.data(), d_values, numKeys));

  // Build the tree
  GpuTimer timer;
  timer.timerStart();
  btree.insertKeys(d_keys, d_values, numKeys, SourceT::DEVICE);
  timer.timerStop();

  printf("Build: %i pairs in %f ms (%0.2f MKeys/sec)\n",
         numKeys,
         timer.getMsElapsed(),
         float(numKeys) * 1e-6 / timer.getSElapsed());

  // cleanup
  hipFree(d_keys);
  hipFree(d_values);
  btree.free();
  return 0;
}